#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <array>
#include <string>
#include "config.h"
#include <vector>
#include <openssl/md5.h>
#include "md5.cuh"
#include <time.h>
using namespace std;
#define MD5_BLOCK_SIZE 16  // MD5 produces a 16-byte hash

vector<string> results;  // Store permutations here

// Helper function for backtracking with fixed length
void backtrack(const vector<char>& chars, string& current, int maxLength, vector<string>& results) {
    // Base case: if the current permutation reaches maxLength, print it
    if (current.size() == maxLength) {
        results.push_back(current);
        return;
    }

    // Recur with each character in the character set
    for (char c : chars) {
        current.push_back(c);  // Add character to the current permutation
        backtrack(chars, current, maxLength, results);  // Recurse
        current.pop_back();  // Remove character to backtrack
    }
}
// Function to start generating permutations with a fixed length
void generatePermutations(const vector<char>& chars, int length, vector<string>& results, char c) {
    string current(1, c);  // Holds the current permutation
    backtrack(chars, current, length, results);
}

extern void mcm_cuda_md5_hash_batch(BYTE* in, WORD inlen, BYTE* out, WORD n_batch);

void bruteforce(unsigned char targetDigests[][MD5_DIGEST_LENGTH], int numTargets, char* guessPool, int guessPoolSize, int length){
      // Allocate device memory for output data (MD5 hashes)
    BYTE* cuda_outdata;
    hipMallocManaged(&cuda_outdata, MD5_BLOCK_SIZE * guessPoolSize * sizeof(BYTE));

    // Launch the batch MD5 hashing function
    WORD batch_size = guessPoolSize;  // Process all guesses in one batch
    mcm_cuda_md5_hash_batch(reinterpret_cast<BYTE*>(guessPool), length, cuda_outdata, batch_size);

    // Check the MD5 hashes for a match with the target digest
    for (int i = 0; i < guessPoolSize; ++i) {
        BYTE* digest = cuda_outdata + i * MD5_BLOCK_SIZE;  // Retrieve the ith digest from cuda_outdata

        // Check the current digest against each target digest
        for (int j = 0; j < numTargets; ++j) {
            if (memcmp(targetDigests[j], digest, MD5_BLOCK_SIZE) == 0) {
                // Assuming `guessPool[i]` holds the permutation that produced this digest
                std::cout << "Hash matched for password \"";

                for (int l = 0; l < length; ++l) {
                    std::cout << guessPool[i * length + l];
                }

                std::cout << "\" with hash: ";
                for (int k = 0; k < MD5_BLOCK_SIZE; ++k) {
                    printf("%02x", targetDigests[j][k]);
                }
                std::cout << std::endl;
            }
        }
    }

    // Clean up the allocated device memory
    hipFree(cuda_outdata);
}



int main() {

    const int numTargets = 10;  // Number of target passwords
    const char* targetPasswords[] = {"YVPSW", "NUTRT", "PWUJX", "LGSAG", "XLRYL", "HGGDU", "VDLCT", "BHKNU", "CFJHM", "VUBEM"};
    unsigned char targetDigests[numTargets][MD5_DIGEST_LENGTH];  // Array to store MD5 digests for each target password

    for (int i = 0; i < numTargets; ++i) {
        MD5((const unsigned char*)targetPasswords[i], strlen(targetPasswords[i]), targetDigests[i]);
    }

    // Output MD5 digests for verification (in hexadecimal format)
    for (int i = 0; i < numTargets; ++i) {
        std::cout << "MD5 digest for \"" << targetPasswords[i] << "\": ";
        for (int j = 0; j < MD5_DIGEST_LENGTH; ++j) {
            printf("%02x", targetDigests[i][j]);
        }
        std::cout << std::endl;
    }

    
    // Initialize chars vector with 'A'-'Z'
    vector<char> chars;
    for (char c = 'A'; c <= 'Z'; ++c) {
        chars.push_back(c);
    }
    


    vector<string> results;  // Store permutations here
    // Generate and print permutations of the specified length

    clock_t start_time;
    clock_t end_time;
    double elapsed_time;

    int length = 5;
    char* guessPool;
    int guessPoolSize;
    for (char c : chars) {
       
        // Generate permutations of the specified length
        generatePermutations(chars, length, results, c);

        // Clear permutations for the next iteration
        guessPoolSize = results.size();
        hipMallocManaged(&guessPool, guessPoolSize * length * sizeof(char));

        for (int i = 0; i < guessPoolSize; i++) {
            memcpy(guessPool + i * length, results[i].c_str(), length * sizeof(char));
        }
        start_time = clock();
        // Apply brute force to check permutations against all target digests
        bruteforce(targetDigests, numTargets, guessPool, guessPoolSize, length);
        end_time = clock();
        elapsed_time = elapsed_time + (double)(end_time - start_time) / CLOCKS_PER_SEC;

        hipFree(guessPool);
        
        results.clear();
    }
    printf("The program took %.6f seconds to execute.\n", elapsed_time);
    return 0;
}